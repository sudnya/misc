

#include <hip/hip_runtime.h>
#include <vector>
#include <cassert>
#include <iostream>

__device__ int dummy;

__global__ void dummy_kernel()
{
    dummy = 0;
}

void check(hipError_t status)
{
    assert(status == hipSuccess);
}

void benchmark(int streamCount, int kernelsPerStream)
{
    std::vector<hipStream_t> streams;
    std::vector<hipEvent_t> beginEvents;
    std::vector<hipEvent_t> endEvents;

    for(int i = 0; i < streamCount; ++i)
    {
        streams.push_back(nullptr);
        check(hipStreamCreateWithFlags(&streams.back(), hipStreamNonBlocking));
    }

    for(int i = 0; i < streamCount; ++i)
    {
        endEvents.push_back(nullptr);
        check(hipEventCreate(&endEvents.back()));
        beginEvents.push_back(nullptr);
        check(hipEventCreate(&beginEvents.back()));
    }

    for(int i = 0; i < streamCount; ++i)
    {
        check(hipEventRecord(beginEvents[i], streams[i]));
    }

    for(int i = 0; i < kernelsPerStream; ++i)
    {
        for(int j = 0; j < streamCount; ++j)
        {
            dummy_kernel<<<1, 1, 0, streams[j]>>>();
        }
    }

    for(int i = 0; i < streamCount; ++i)
    {
        check(hipEventRecord(endEvents[i], streams[i]));
    }

    check(hipDeviceSynchronize());

    double duration = 0.0;

    for(int i = 0; i < streamCount; ++i)
    {
        float ms = 0.0f;

        check(hipEventElapsedTime(&ms, beginEvents[i], endEvents[i]));

        duration = std::max(static_cast<double>(ms), duration);
    }

    std::cout << "Overhead per kernel is " << (duration * 1e3) / (streamCount * kernelsPerStream) << "us\n";

    for(auto event : beginEvents)
    {
        check(hipEventDestroy(event));
    }

    for(auto event : endEvents)
    {
        check(hipEventDestroy(event));
    }

    for(auto stream : streams)
    {
        check(hipStreamDestroy(stream));
    }

}

int main()
{
    benchmark(1, 100);

    benchmark(1, 50*100000);
    benchmark(50, 100000);

    return 0;
}



