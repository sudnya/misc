/***********************************************************
#
# \file    benchmark_px_conv2d.cu
# \author  Sudnya Padalikar <mailsudnya@gmail.co>
# \date    Sudnya July 3, 2016
# \brief   benchmark for cudnn - with and without per
#          sample filters
***********************************************************/

#include <hipDNN.h>
#include <string>
#include <iostream>
#include <vector>
#include <sstream>
#include <stdexcept>

class BenchmarkOptions
{
    public:
        BenchmarkOptions(
            int imageW,
            int imageH,
            int imageChannels,
            int miniBatchSize,

            int filterW,
            int filterH,
            int filters,

            int iterations,
            float alpha,
            float beta,

            bool usePerExampleFilters,

            hipdnnConvolutionFwdAlgo_t algorithm)
        :
            imageW(imageW),
            imageH(imageH),
            imageChannels(imageChannels),
            miniBatchSize(miniBatchSize),
            filterW(filterW),
            filterH(filterH),
            filters(filters),
            iterations(iterations),
            alpha(alpha),
            beta(beta),
            usePerExampleFilters(usePerExampleFilters),
            algorithm(algorithm)
        {}

    public:
        int imageW;
        int imageH;
        int imageChannels;
        int miniBatchSize;

        int filterW;
        int filterH;
        int filters;

        int iterations;
        float alpha;
        float beta;

        bool usePerExampleFilters;

        hipdnnConvolutionFwdAlgo_t algorithm;

};

void check(hipError_t status)
{
    if(status != hipSuccess)
    {
        throw std::runtime_error("CUDA function failed.");
    }
}

void check(hipdnnStatus_t status)
{
    if(status != HIPDNN_STATUS_SUCCESS)
    {
        throw std::runtime_error("CUDNN function failed");
    }
}

std::string toString(const BenchmarkOptions& options)
{
    std::stringstream stream;

    stream << "cudnnForwardConvolution(input {" << options.imageH << ", " << options.imageW << ", " << options.imageChannels << ", "
           << options.miniBatchSize << "}, filter {" << options.filterW << ", " << options.filterH << ", " << options.imageChannels << ", " << options.filters << "},"
           << ")";

    return stream.str();
}

void runBenchmark(const BenchmarkOptions& options)
{
    float* source = nullptr;
    float* filter = nullptr;
    float* destination = nullptr;

    int filterMultiplier = 1;
    int miniBatchFactor = options.miniBatchSize;

    if(options.usePerExampleFilters)
    {
        filterMultiplier = options.miniBatchSize;
        miniBatchFactor = 1;
    }

    check(hipMalloc(&source, sizeof(float) * options.miniBatchSize * options.imageChannels * options.imageH * options.imageW));
    check(hipMalloc(&destination, sizeof(float) * options.miniBatchSize * options.filters * options.imageH * options.imageW));
    check(hipMalloc(&filter, sizeof(float) * filterMultiplier * options.filters * options.imageChannels * options.filterH * options.filterW));

    // create descriptors and allocate memory
    hipdnnHandle_t handle;

    check(hipdnnCreate(&handle));

    hipdnnTensorDescriptor_t sourceDescriptor;

    check(hipdnnCreateTensorDescriptor(&sourceDescriptor));

    check(hipdnnSetTensor4dDescriptor(sourceDescriptor,
                                     HIPDNN_TENSOR_NCHW,
                                     HIPDNN_DATA_FLOAT,
                                     miniBatchFactor,
                                     options.imageChannels,
                                     options.imageH,
                                     options.imageW));

    hipdnnTensorDescriptor_t destinationDescriptor;

    check(hipdnnCreateTensorDescriptor(&destinationDescriptor));

    check(hipdnnSetTensor4dDescriptor(destinationDescriptor,
                                     HIPDNN_TENSOR_NCHW,
                                     HIPDNN_DATA_FLOAT,
                                     miniBatchFactor,
                                     options.filters,
                                     options.imageH,
                                     options.imageW));

    hipdnnFilterDescriptor_t filterDescriptor;

    check(hipdnnCreateFilterDescriptor(&filterDescriptor));


    check(hipdnnSetFilter4dDescriptor(filterDescriptor,
                                     HIPDNN_DATA_FLOAT, // image data type
                                     HIPDNN_TENSOR_NCHW,
                                     options.filters,        // number of output feature maps
                                     options.imageChannels,        // number of input feature maps
                                     options.filterH,        // height of each input filter
                                     options.filterW         // width of  each input fitler
                                     ));

    hipdnnConvolutionDescriptor_t convolutionDescriptor;

    check(hipdnnCreateConvolutionDescriptor(&convolutionDescriptor));

    check(hipdnnSetConvolution2dDescriptor(convolutionDescriptor,
                                          options.filterH / 2,    // zero-padding height
                                          options.filterW / 2,    // zero-padding width
                                          1,        // vertical filter stride
                                          1,        // horizontal filter stride
                                          1, // upscale the input in x-direction
                                          1, // upscale the input in y-direction
                                          HIPDNN_CONVOLUTION));

    size_t workspaceSize = 0;

    check(hipdnnGetConvolutionForwardWorkspaceSize(handle,
                                                  sourceDescriptor,
                                                  filterDescriptor,
                                                  convolutionDescriptor,
                                                  destinationDescriptor,
                                                  options.algorithm,
                                                  &workspaceSize));

    void* workspace;
    check(hipMalloc(&workspace, workspaceSize));

    // start timers
    hipEvent_t start;
    hipEvent_t end;

    hipStream_t stream;

    check(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

    check(hipdnnSetStream(handle, stream));

    // warm start
    check(hipdnnConvolutionForward(handle,
                                  &options.alpha,
                                  sourceDescriptor,
                                  source,
                                  filterDescriptor,
                                  filter,
                                  convolutionDescriptor,
                                  options.algorithm,
                                  workspace,
                                  workspaceSize,
                                  &options.beta,
                                  destinationDescriptor,
                                  destination));

    check(hipDeviceSynchronize());

    check(hipEventCreate(&start));
    check(hipEventCreate(&end));

    check(hipEventRecord(start, stream));

    // run conv in a loop
    for(size_t i = 0; i < options.iterations; ++i)
    {
        for(size_t j = 0; j < filterMultiplier; ++j)
        {
            size_t inputOffset  = j * (options.imageH * options.imageW * options.imageChannels);
            size_t filterOffset = j * (options.filterH * options.filterW * options.imageChannels * options.filters);
            size_t outputOffset = inputOffset;

            check(hipdnnConvolutionForward(handle,
                                          &options.alpha,
                                          sourceDescriptor,
                                          source + inputOffset,
                                          filterDescriptor,
                                          filter + filterOffset,
                                          convolutionDescriptor,
                                          options.algorithm,
                                          workspace,
                                          workspaceSize,
                                          &options.beta,
                                          destinationDescriptor,
                                          destination + outputOffset));
        }
    }

    // stop timers
    check(hipEventRecord(end, stream));

    check(hipDeviceSynchronize());

    // print status
    float ms = 0.0f;

    check(hipEventElapsedTime(&ms, start, end));

    std::cout << toString(options) << " completed in " << (1.e3*ms / options.iterations) << "us\n";

    // free memory

    check(hipEventDestroy(start));
    check(hipEventDestroy(end));

    check(hipStreamDestroy(stream));

    check(hipdnnDestroyConvolutionDescriptor(convolutionDescriptor));
    check(hipdnnDestroyFilterDescriptor(filterDescriptor));
    check(hipdnnDestroyTensorDescriptor(destinationDescriptor));
    check(hipdnnDestroyTensorDescriptor(sourceDescriptor));
    check(hipdnnDestroy(handle));

    check(hipFree(workspace));
    check(hipFree(source));
    check(hipFree(destination));
    check(hipFree(filter));
}

std::vector<BenchmarkOptions> createOptions(int miniBatchSize, int iterations)
{
    std::vector<BenchmarkOptions> options;

    options.emplace_back(256, 256, 3, miniBatchSize, 3, 3, 64, iterations, 1.0, 0.0, false, HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM);
    options.emplace_back(256, 256, 3, miniBatchSize, 3, 3, 64, iterations, 1.0, 0.0, true, HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM);

    options.emplace_back(128, 128, 64, miniBatchSize, 3, 3, 64, iterations, 1.0, 0.0, false, HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM);
    options.emplace_back(128, 128, 64, miniBatchSize, 3, 3, 64, iterations, 1.0, 0.0, true, HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM);

    options.emplace_back(64, 64, 64, miniBatchSize, 3, 3, 128, iterations, 1.0, 0.0, false, HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM);
    options.emplace_back(64, 64, 64, miniBatchSize, 3, 3, 128, iterations, 1.0, 0.0, true, HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM);

    options.emplace_back(32, 32, 128, miniBatchSize, 3, 3, 128, iterations, 1.0, 0.0, false, HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM);
    options.emplace_back(32, 32, 128, miniBatchSize, 3, 3, 128, iterations, 1.0, 0.0, true, HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM);

    options.emplace_back(16, 16, 128, miniBatchSize, 3, 3, 256, iterations, 1.0, 0.0, false, HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM);
    options.emplace_back(16, 16, 128, miniBatchSize, 3, 3, 256, iterations, 1.0, 0.0, true, HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM);

    options.emplace_back(8, 8, 256, miniBatchSize, 3, 3, 256, iterations, 1.0, 0.0, false, HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM);
    options.emplace_back(8, 8, 256, miniBatchSize, 3, 3, 256, iterations, 1.0, 0.0, true, HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM);

    options.emplace_back(4, 4, 256, miniBatchSize, 3, 3, 256, iterations, 1.0, 0.0, false, HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM);
    options.emplace_back(4, 4, 256, miniBatchSize, 3, 3, 256, iterations, 1.0, 0.0, true, HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM);

    return options;
}

void runBenchmarks()
{
    auto options = createOptions(64, 100);

    for(auto& option : options)
    {
        runBenchmark(option);
    }
}

int main()
{
    runBenchmarks();

    return 0;
}



